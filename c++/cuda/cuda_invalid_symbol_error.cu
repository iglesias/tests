/*
 * cuda_innvalid_symbol_error.cu
 *
 * Compilation triggering runtime error:
 *  nvcc -gencode arch=compute_52,code=sm_52 -std=c++11 cuda_invalid_symbol_error.cu -o cuda_invalid_symbol_error
 *
 * Error-free compilation:
 *  nvcc -arch=sm_52 -std=c++11 cuda_invalid_symbol_error.cu -o cuda_invalid_symbol_error
 *
 * http://stackoverflow.com/questions/42813955/invalid-device-symbol-error-depending-on-nvcc-flags
 */


#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>

// Matrix side size (they are square).
const int N = 3;
const int num_mats = 14;

// Rotation matrices.
__constant__ float rot_mats_device[num_mats * N * N];

int main() {
  auto errSetDevice = hipSetDevice(0);
  if (errSetDevice != hipSuccess) {
    std::cout << "SetDevice error: " << hipGetErrorString(errSetDevice)
              << std::endl;
  }

  std::vector<float> rot_mats_host(num_mats * N * N);
  for (int i = 0; i < rot_mats_host.size(); i++) rot_mats_host[i] = i;

  auto errMemcpyToSymbol = hipMemcpyToSymbol(
      HIP_SYMBOL(rot_mats_device), rot_mats_host.data(), sizeof(rot_mats_device));

  if (errMemcpyToSymbol != hipSuccess) {
    std::cout << "MemcpyToSymbol error: "
              << hipGetErrorString(errMemcpyToSymbol) << std::endl;
  }
}
