/*
 * /usr/local/cuda-8.0/bin/nvcc -Xcompiler -Wall -O3 -arch sm_61 -std=c++11 -o use_shared use_shared.cu
 *
 * We have N (in the order of 30_000) real numbers and M entities (in the order
 * of 2_000_000). We have associations between entities and numbers (that is,
 * an entity is associated with some of the N numbers).  The average number of
 * associations for an entity is in the order of 10, with an upper bound equal
 * to MAX_M2N, around 100).  The N numbers in are in the array 'numbers' and
 * the associations are in the array 'map'. 'map' is an array of M lists, the
 * lists are stored in fixed-size arrays and contain indices to the array
 * 'numbers' (-1 is used for sentinel values trailing in the arrays).
 *
 * We want to build an array of M real numbers where each element indexed by m
 * is obtained by the product of the elements in 'numbers' in the list map[m].
 * A picture is worth a thousand words:
 *
 * Suppose the array of size N is in general numbers[0] numbers[1] ... numbers[N-1]
 *
 * and an example of the map is
 *
 * m=0   10 67 13 -1 -1 -1 ...
 * m=1   0 -1 -1 -1 -1 -1 ....
 * m=2   1 5 10 23 ....
 * ...   ......
 *
 * then the three first elements of the desired array are given by
 *
 *    result[0] = numbers[10] * numbers[67] * numbers[13]
 *    result[1] = numbers[0]
 *    result[2] = numbers[1] * numbers[5] * numbers[10] * numbers[23] * ...
 *
 * This program implements this algorithm in the gpu with cuda. Two kernels are
 * provided: one of them leverages shared memory to cache the map while the
 * other reads it directly from global memory when making the computations.
 *
 * Very interestingly, we note that the performance gap between the kernels w/
 * and w/o shared memory depends on the variance of the distribution of the
 * number of associations (see num_associations_dis). When the number of
 * associations is constant (drawn from a uniform [MAX_M2N, MAX_M2N]), the
 * kernel w/ shared memory is best. On the other hand, when a uniform [0,
 * MAX_M2N] is used, the kernel w/o shared memory performs better.
 *
 */


#include <hip/hip_runtime.h>
#include <algorithm>
#include <array>
#include <chrono>
#include <cmath>
#include <cstring>
#include <iostream>
#include <random>

#define CUDA_CALL(F) if ((F) != hipSuccess) { printf("Cuda call error %s at %s:%d\n", hipGetErrorString(hipGetLastError()), __FILE__, __LINE__); abort(); }
#define CUDA_CHECK_CALL() { auto err = hipGetLastError(); if (err != hipSuccess) { printf("Cuda check call error %s at %s:%d\n", hipGetErrorString(err), __FILE__, __LINE__-1); abort(); } }

using real = float;

const short N = 32768 - 1;
const size_t M = 1024 * 2048;
const unsigned char MAX_M2N = 80;
const size_t BLOCK_DIM = 256;

std::default_random_engine gen;

std::array<real, N> numbers;
real* dev_numbers;

std::array<short, M * MAX_M2N> map;
short* dev_map;

real* dev_result;
real* dev_result_with_shared;

std::array<real, M> result;
std::array<real, M> result_with_shared;

void FillNumbers()
{
  std::uniform_real_distribution<real> dis{0, 1};
  std::for_each(numbers.begin(), numbers.end(), [&dis](real& number) { number = dis(gen); });
}

void FillMap()
{
  std::memset(map.data(), -1, map.size() * sizeof(short));

  std::uniform_int_distribution<int> num_associations_dis{0, MAX_M2N};
  std::uniform_int_distribution<short> associations_dis{0, N-1};

  for (size_t i = 0; i < M; i++) {
    unsigned char num_associations = num_associations_dis(gen);
    std::for_each(map.begin() + i * MAX_M2N, map.begin() + i * MAX_M2N + num_associations,
                  [&associations_dis](short& item) { item = associations_dis(gen); });
  }
}

__global__ void kernel(real* dev_result, const real* dev_numbers, const short* dev_map)
{
  const int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < M) {
    real val = 1.0;
    const short* map = dev_map + idx * MAX_M2N;
    for (unsigned char i = 0; i < MAX_M2N; i++) {
      short item = *map++;
      if (item == -1) break;
      val *= dev_numbers[item];
    }

    dev_result[idx] = val;
  }
}

__global__ void kernel_with_shared(real* dev_result_with_shared, const real* dev_numbers, const short* dev_map)
{
  const int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < M) {
    __shared__ short map_cache[BLOCK_DIM * MAX_M2N];
    const short* map = dev_map + idx * MAX_M2N;
    for (unsigned char i = 0; i < MAX_M2N; i++) {
      short item = *map++;
      map_cache[threadIdx.x * MAX_M2N + i] = item;
      if (item == -1) break;
    }

    __syncthreads();

    real val = 1.0;
    for (unsigned char i = 0; i < MAX_M2N; i++) {
      short item = map_cache[threadIdx.x * MAX_M2N + i];
      if (item == -1) break;
      val *= dev_numbers[item];
    }

    dev_result_with_shared[idx] = val;
  }
}

int main()
{
  {
    std::random_device r;
    gen = std::default_random_engine(r());
  }

  FillNumbers();
  FillMap();

  CUDA_CALL(hipMalloc(&dev_numbers, N * sizeof(real)));
  CUDA_CALL(hipMalloc(&dev_map, M * MAX_M2N * sizeof(short)));
  CUDA_CALL(hipMalloc(&dev_result, M * sizeof(real)));
  CUDA_CALL(hipMalloc(&dev_result_with_shared, M * sizeof(real)));

  CUDA_CALL(hipMemcpy(dev_numbers, numbers.data(), N * sizeof(real), hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(dev_map, map.data(), M * MAX_M2N * sizeof(short), hipMemcpyHostToDevice));

  const int num_threads = BLOCK_DIM;
  const int num_blocks = M / num_threads;

  auto kernel_start = std::chrono::high_resolution_clock::now();
  for (int i = 0; i < 100; i++) {
    kernel<<<num_blocks, num_threads>>>(dev_result, dev_numbers, dev_map);
    CUDA_CHECK_CALL();
    CUDA_CALL(hipDeviceSynchronize());
  }
  auto kernel_finish = std::chrono::high_resolution_clock::now();

  for (int i = 0; i < 100; i++) {
    kernel_with_shared<<<num_blocks, num_threads>>>(dev_result_with_shared, dev_numbers, dev_map);
    CUDA_CHECK_CALL();
    CUDA_CALL(hipDeviceSynchronize());
  }
  auto kernel_with_shared_finish = std::chrono::high_resolution_clock::now();

  CUDA_CALL(hipMemcpy(result.data(), dev_result, M * sizeof(real), hipMemcpyDeviceToHost));
  CUDA_CALL(hipMemcpy(result_with_shared.data(), dev_result_with_shared, M * sizeof(real), hipMemcpyDeviceToHost));

  double diff = 0;
  for (size_t i = 0; i < result.size(); i++) diff += std::abs(result[i] - result_with_shared[i]);
  std::cout << "Result difference: " << diff << '.' << std::endl;

  std::cout << std::chrono::duration_cast<std::chrono::milliseconds>(kernel_finish - kernel_start).count() << ' '
            << std::chrono::duration_cast<std::chrono::milliseconds>(kernel_with_shared_finish - kernel_finish).count() << '\n';

  CUDA_CALL(hipFree(dev_numbers));
  CUDA_CALL(hipFree(dev_map));
  CUDA_CALL(hipFree(dev_result));
  CUDA_CALL(hipFree(dev_result_with_shared));
}
